#include "hip/hip_runtime.h"
/* ------------------------------------------------------------------------------------
CUDA code for calculating the policy function

Coder: Hyok Jung Kim

This code is CUDA version of ProdTents() in AdaptiveSparseGrid.h
------------------------------------------------------------------------------------ */
#include "ProdTents.cuh"
#include "stdio.h"

using namespace std;

__global__ void ProdTentsKernel(const double *xx_coordinate,
	const int *in_idxlvl, const int NS, const int Niter,
	double *out, const double *inParams) {

	int row = blockIdx.x*blockDim.x + threadIdx.x;
	
	// Evaluate rows that are relevant
	if (row < Niter) {
		int idxrow = row*NS * 2;
		double h_l = 2.0;
		double tempval = 1.0;

		out[row] = 1;

		#pragma unroll
		for (int ii = 0; ii < NS; ii++) {
			h_l = 2.0;

			#pragma unroll
			for (int jj = 0; jj < in_idxlvl[idxrow + ii + NS]; jj++) { h_l /= 2.0; }

			tempval = fmin(1.0 - (xx_coordinate[ii] - h_l*in_idxlvl[idxrow + ii]) / h_l, 1.0 + (xx_coordinate[ii] - h_l*in_idxlvl[idxrow + ii]) / h_l);
			tempval = fmax(tempval, 0.0);
			out[row] *= tempval;
		}
		out[row] *= inParams[row];
	}
}

__declspec(dllexport) void ProdTentsHost(const double *xx_coordinate, const int *in_idxlvl,
	const int NS, const double *inParams, const int Niter, double *out_vec){

	size_t xx_memsize = NS * sizeof(double);
	size_t param_memsize = Niter * sizeof(double);
	size_t matsize = NS * Niter * 2 * sizeof(int);
	// hipError_t error;
	/* For future use to check errors
	if (error != hipSuccess)
	{
		printf("Error : %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}
	*/

	/*
		(1) Pass grid point
	*/
	double *xx_pass;
	hipMalloc(&xx_pass, xx_memsize);
	hipMemcpy(xx_pass, xx_coordinate, xx_memsize, hipMemcpyHostToDevice);

	/*
		(2) Pass indexes and levels as matrices
	*/
	int *idxlvl_pass;
	hipMalloc(&idxlvl_pass, matsize);
	hipMemcpy(idxlvl_pass, in_idxlvl, matsize, hipMemcpyHostToDevice);

	/*
		(3) Pass parameters
	*/
	double *param_pass;
	hipMalloc(&param_pass, param_memsize);
	hipMemcpy(param_pass, inParams, param_memsize, hipMemcpyHostToDevice);
	
	/*
		(4) Out memory for the DEVICE
	*/
	double *out_pass;
	hipMalloc(&out_pass, param_memsize);

	/*
		Set Dimensions
			- I assume that we have 1024 cores per each grid
	*/
	int ngridx = static_cast<int>(ceil((double)Niter / 1024.0));

	/*
		Evaluate
	*/
	ProdTentsKernel<<<ngridx, 1024>>>(xx_pass, idxlvl_pass, NS, Niter, out_pass, param_pass);
	
	/*
		Copy back to the Host memory
	*/
	hipMemcpy(out_vec, out_pass, param_memsize, hipMemcpyDeviceToHost);

	/*
		Free Device memory
	*/
	hipFree(xx_pass); hipFree(idxlvl_pass);
	hipFree(param_pass); hipFree(out_pass);
}